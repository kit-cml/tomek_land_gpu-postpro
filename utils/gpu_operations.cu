#include <hip/hip_runtime.h>

#include <cstdio>

#include "gpu_operations.hpp"
#include "constants.hpp"

/**
 * @brief Prepares GPU memory space and copies initial data from host to device.
 *
 * @param d_ALGEBRAIC Pointer to device memory for algebraic variables.
 * @param Tomek_num_of_algebraic Number of algebraic variables.
 * @param sample_size Number of samples.
 * @param d_CONSTANTS Pointer to device memory for constants.
 * @param Tomek_num_of_constants Number of constants.
 * @param d_RATES Pointer to device memory for rates.
 * @param Tomek_num_of_rates Number of rates.
 * @param d_STATES Pointer to device memory for states.
 * @param Tomek_num_of_states Number of states.
 * @param d_p_param Pointer to device memory for parameters.
 * @param temp_result Pointer to device memory for temporary results.
 * @param cipa_result Pointer to device memory for CIPA results.
 * @param d_STATES_RESULT Pointer to device memory for states results.
 * @param d_ic50 Pointer to device memory for IC50 data.
 * @param ic50 Pointer to host memory for IC50 data.
 * @param d_conc Pointer to device memory for concentration data.
 * @param conc Pointer to host memory for concentration data.
 * @param p_param Pointer to host memory for parameters.
 */
void prepingGPUMemory(int sample_size, double *&d_ALGEBRAIC, double *&d_CONSTANTS, double *&d_RATES, double *&d_STATES,
                      double *&d_mec_ALGEBRAIC, double *&d_mec_CONSTANTS, double *&d_mec_RATES, double *&d_mec_STATES,
                      param_t *&d_p_param, cipa_t *&temp_result, cipa_t *&cipa_result, double *&d_STATES_RESULT, double *&d_ic50, 
                      
                      double *ic50, double *&d_conc, double *conc, param_t *p_param) {
    printf("preparing GPU memory space \n");

    // Allocate memory on the device
    hipMalloc(&d_ALGEBRAIC, Tomek_num_of_algebraic * sample_size * sizeof(double));
    hipMalloc(&d_CONSTANTS, Tomek_num_of_constants * sample_size * sizeof(double));
    hipMalloc(&d_RATES, Tomek_num_of_rates * sample_size * sizeof(double));
    hipMalloc(&d_STATES, Tomek_num_of_states * sample_size * sizeof(double));
    hipMalloc(&d_mec_ALGEBRAIC, Land_num_of_algebraic * sample_size * sizeof(double));
    hipMalloc(&d_mec_CONSTANTS, Land_num_of_constants * sample_size * sizeof(double));
    hipMalloc(&d_mec_RATES, Land_num_of_rates * sample_size * sizeof(double));
    hipMalloc(&d_mec_STATES, Land_num_of_states * sample_size * sizeof(double));
    hipMalloc(&d_p_param, sizeof(param_t));
    hipMalloc(&temp_result, sample_size * sizeof(cipa_t));
    hipMalloc(&cipa_result, sample_size * sizeof(cipa_t));
    hipMalloc(&d_STATES_RESULT, Tomek_num_of_states * sample_size * sizeof(double));

    // Allocate memory for IC50 and concentration data
    hipMalloc(&d_ic50, sample_size * 14 * sizeof(double));
    hipMalloc(&d_conc, sample_size * sizeof(double));

    // Copy data from host to device
    printf("Copying sample files to GPU memory space \n");
    hipMemcpy(d_ic50, ic50, sample_size * 14 * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_conc, conc, sample_size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_p_param, p_param, sizeof(param_t), hipMemcpyHostToDevice);
}

void prepingGPUMemoryPostpro(int sample_size, double *&d_ALGEBRAIC, double *&d_CONSTANTS, double *&d_RATES, double *&d_STATES, double *d_STATES_cache,
                      double *&d_mec_ALGEBRAIC, double *&d_mec_CONSTANTS, double *&d_mec_RATES, double *&d_mec_STATES,
                      param_t *&d_p_param, cipa_t *&temp_result, cipa_t *&cipa_result, double *&d_STATES_RESULT, double *&d_ic50, 
                     
                      double *ic50, double *&d_conc, double *conc, param_t *p_param, double *cache,
                      double *time, double *dt, double *states, double *ical, double *inal, double *cai_result, double *ina, double *ito, double *ikr, double *iks, double *ik1, double *tension) {
    printf("preparing GPU memory space \n");

    // Allocate memory on the device
    hipMalloc(&d_ALGEBRAIC, Tomek_num_of_algebraic * sample_size * sizeof(double));
    hipMalloc(&d_CONSTANTS, Tomek_num_of_constants * sample_size * sizeof(double));
    hipMalloc(&d_RATES, Tomek_num_of_rates * sample_size * sizeof(double));
    hipMalloc(&d_STATES, Tomek_num_of_states * sample_size * sizeof(double));
    hipMalloc(&d_STATES_cache, (Tomek_num_of_states + 2) * sample_size * sizeof(double));

    hipMalloc(&d_mec_ALGEBRAIC, Land_num_of_algebraic * sample_size * sizeof(double));
    hipMalloc(&d_mec_CONSTANTS, Land_num_of_constants * sample_size * sizeof(double));
    hipMalloc(&d_mec_RATES, Land_num_of_rates * sample_size * sizeof(double));
    hipMalloc(&d_mec_STATES, Land_num_of_states * sample_size * sizeof(double));

    hipMalloc(&d_p_param, sizeof(param_t));
    hipMalloc(&temp_result, sample_size * sizeof(cipa_t));
    hipMalloc(&cipa_result, sample_size * sizeof(cipa_t));
    hipMalloc(&d_STATES_RESULT, Tomek_num_of_states * sample_size * sizeof(double)); // check for wat later

        hipMalloc(&time, sample_size * datapoint_size * sizeof(double));
        hipMalloc(&dt, sample_size * datapoint_size * sizeof(double));
        hipMalloc(&states, sample_size * datapoint_size * sizeof(double));
        hipMalloc(&ical, sample_size * datapoint_size * sizeof(double));
        hipMalloc(&inal, sample_size * datapoint_size * sizeof(double));
        hipMalloc(&cai_result, sample_size * datapoint_size * sizeof(double));
        hipMalloc(&ina, sample_size * datapoint_size * sizeof(double));
        hipMalloc(&ito, sample_size * datapoint_size * sizeof(double));
        hipMalloc(&ikr, sample_size * datapoint_size * sizeof(double));
        hipMalloc(&iks, sample_size * datapoint_size * sizeof(double));
        hipMalloc(&ik1, sample_size * datapoint_size * sizeof(double));
        hipMalloc(&tension, sample_size * datapoint_size * sizeof(double));

   printf("Copying sample files to GPU memory space \n");
   hipMalloc(&d_ic50, sample_size * 14 * sizeof(double));
//    hipMalloc(&d_cvar, sample_size * 18 * sizeof(double));
   hipMalloc(&d_conc, sample_size * sizeof(double));

   hipMemcpy(d_STATES_cache, cache, (Tomek_num_of_states + 2) * sample_size * sizeof(double), hipMemcpyHostToDevice);
   hipMemcpy(d_ic50, ic50, sample_size * 14 * sizeof(double), hipMemcpyHostToDevice);
        
//    hipMemcpy(d_cvar, cvar, sample_size * 18 * sizeof(double), hipMemcpyHostToDevice);
   hipMemcpy(d_conc, conc, sample_size * sizeof(double), hipMemcpyHostToDevice);
   hipMemcpy(d_p_param, p_param, sizeof(param_t), hipMemcpyHostToDevice);
}

/**
 * @brief Frees allocated memory on both the host and device.
 *
 * @param d_ALGEBRAIC Pointer to device memory for algebraic variables.
 * @param d_CONSTANTS Pointer to device memory for constants.
 * @param d_RATES Pointer to device memory for rates.
 * @param d_STATES Pointer to device memory for states.
 * @param d_p_param Pointer to device memory for parameters.
 * @param temp_result Pointer to device memory for temporary results.
 * @param cipa_result Pointer to device memory for CIPA results.
 * @param d_STATES_RESULT Pointer to device memory for states results.
 * @param d_ic50 Pointer to device memory for IC50 data.
 * @param ic50 Pointer to host memory for IC50 data.
 * @param conc Pointer to host memory for concentration data.
 * @param h_states Pointer to host memory for states.
 * @param h_cipa_result Pointer to host memory for CIPA results.
 * @param p_param Pointer to host memory for parameters.
 */
void freeingMemory(double *d_ALGEBRAIC, double *d_CONSTANTS, double *d_RATES, double *d_STATES, double *d_mec_ALGEBRAIC,
                   double *d_mec_CONSTANTS, double *d_mec_RATES, double *d_mec_STATES, param_t *d_p_param,
                   cipa_t *temp_result, cipa_t *cipa_result, double *d_STATES_RESULT, double *d_ic50, double *ic50,
                   double *conc, double *h_states, cipa_t *h_cipa_result, param_t *p_param) {
    // Free GPU memory
    hipFree(d_ALGEBRAIC);
    hipFree(d_CONSTANTS);
    hipFree(d_RATES);
    hipFree(d_STATES);
    hipFree(d_mec_ALGEBRAIC);
    hipFree(d_mec_CONSTANTS);
    hipFree(d_mec_RATES);
    hipFree(d_mec_STATES);
    hipFree(d_p_param);
    hipFree(temp_result);
    hipFree(cipa_result);
    hipFree(d_STATES_RESULT);
    hipFree(d_ic50);

    // Free CPU memory
    free(ic50);
    free(conc);
    free(h_states);
    free(h_cipa_result);
    delete p_param;
}

/**
 * @brief Checks the available GPU memory.
 *
 * @param datasize Size of the data to be checked against available GPU memory.
 * @return int 0 if successful, 1 if insufficient memory.
 */
int gpu_check(unsigned int datasize) {
    int num_gpus;
    float percent;
    int id;
    size_t free, total;
    hipGetDeviceCount(&num_gpus);
    for (int gpu_id = 0; gpu_id < num_gpus; gpu_id++) {
        hipSetDevice(gpu_id);
        hipGetDevice(&id);
        hipMemGetInfo(&free, &total);
        percent = (free / (float)total);
        printf("GPU No %d\nFree Memory: %ld, Total Memory: %ld (%f percent free)\n", id, free, total, percent * 100.0);
    }
    percent = 1.0 - (datasize / (float)total);

    return (percent >= 0) ? 0 : 1;
}
