#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// #include "modules/drug_sim.hpp"
#include "modules/cipa_t.cuh"
#include "modules/glob_funct.hpp"
#include "modules/glob_type.hpp"
#include "modules/gpu.cuh"
#include "utils/constants.hpp"
#include "utils/gpu_operations.hpp"
#include "utils/timing.hpp"

#include <algorithm>
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <ctime>
#include <filesystem>
#include <iostream>
#include <math.h>
#include <regex>
#include <string>
#include <sys/stat.h>
#include <unordered_map>
#include <vector>

namespace fs = std::filesystem;

#define ENOUGH ((CHAR_BIT * sizeof(int) - 1) / 3 + 3)
char buffer[255];

// unsigned int datapoint_size = 7000;
const unsigned int sample_limit = 10000;

clock_t START_TIMER;

clock_t tic();
void toc(clock_t start = START_TIMER);

clock_t tic() {
    return START_TIMER = clock();
}

void toc(clock_t start) {
    std::cout
        << "Elapsed time: "
        << (clock() - start) / (double)CLOCKS_PER_SEC << "s"
        << std::endl;
}

int gpu_check(unsigned int datasize) {
    int num_gpus;
    float percent;
    int id;
    size_t free, total;
    hipGetDeviceCount(&num_gpus);
    for (int gpu_id = 0; gpu_id < num_gpus; gpu_id++) {
        hipSetDevice(gpu_id);
        hipGetDevice(&id);
        hipMemGetInfo(&free, &total);
        percent = (free / (float)total);
        printf("GPU No %d\nFree Memory: %ld, Total Memory: %ld (%f percent free)\n", id, free, total, percent * 100.0);
    }
    percent = 1.0 - (datasize / (float)total);
    //// this code strangely gave out too small value, so i disable the safety switch for now

    // printf("The program uses GPU No %d and %f percent of its memory\n", id,percent*100.0);
    // printf("\n");
    // if (datasize<=free) {
    //   return 0;
    // }
    // else {
    //   return 1;
    // }

    return 0;
}

// get the IC50 data from file
drug_t get_IC50_data_from_file(const char *file_name);
// return error and message based on the IC50 data

void addDrugData(char ***arrayOfStrings, int &size, const char newString[]) {
    char **newArray = new char *[size + 1];

    // Copy existing strings to the new array
    for (int i = 0; i < size; ++i) {
        newArray[i] = new char[strlen((*arrayOfStrings)[i]) + 1];
        strcpy(newArray[i], (*arrayOfStrings)[i]);
        delete[] (*arrayOfStrings)[i]; // Deallocate memory for old strings
    }

    // Allocate memory for the new string and copy it
    newArray[size] = new char[strlen(newString) + 1];
    strcpy(newArray[size], newString);

    // Deallocate memory for the old array
    delete[] *arrayOfStrings;

    // Update the pointer to point to the new array
    *arrayOfStrings = newArray;

    // Increment the size
    ++size;
}

int check_IC50_content(const drug_t *ic50, const param_t *p_param);

int get_IC50_data_from_file(const char *file_name, double *ic50) {
    /*
    a host function to take all samples from the file, assuming each sample has 14 features.

    it takes the file name, and an ic50 (already declared in 1D, everything become 1D)
    as a note, the data will be stored in 1D array, means this functions applies flatten.

    it returns 'how many samples were detected?' in integer.
    */
    FILE *fp_drugs;
    //   drug_t ic50;
    char *token;
    char buffer_ic50[255];
    unsigned int idx;

    if ((fp_drugs = fopen(file_name, "r")) == NULL) {
        printf("Cannot open file %s\n",
               file_name);
        return 0;
    }
    idx = 0;
    int sample_size = 0;
    fgets(buffer_ic50, sizeof(buffer_ic50), fp_drugs);                  // skip header
    while (fgets(buffer_ic50, sizeof(buffer_ic50), fp_drugs) != NULL) { // begin line reading
        token = strtok(buffer_ic50, ",");
        while (token != NULL) { // begin data tokenizing
            ic50[idx++] = strtod(token, NULL);
            token = strtok(NULL, ",");
        } // end data tokenizing
        sample_size++;
    } // end line reading

    fclose(fp_drugs);
    return sample_size;
}

int get_IC50_data_from_file(const char *file_name, double *ic50, double *conc, char **drug_name) {
    /*
    a host function to take all samples from the file, assuming each sample has 14 features.

    it takes the file name, and an ic50 (already declared in 1D, everything become 1D)
    as a note, the data will be stored in 1D array, means this functions applies flatten.

    it returns 'how many samples were detected?' in integer.
    */
    FILE *fp_drugs;
    //   drug_t ic50;
    char *token;
    char tmp_drug_name[32];
    char buffer_ic50[255];
    unsigned int idx_ic50, idx_conc;
    int drugsize = 0;

    if ((fp_drugs = fopen(file_name, "r")) == NULL) {
        printf("Cannot open file %s\n",
               file_name);
        return 0;
    }
    idx_ic50 = 0;
    idx_conc = 0;
    int sample_size = 0;
    fgets(buffer_ic50, sizeof(buffer_ic50), fp_drugs);                  // skip header
    while (fgets(buffer_ic50, sizeof(buffer_ic50), fp_drugs) != NULL) { // begin line reading
        /*
        TODO: Extracting token from file
        1. take token for each file
        2. check the first token to drug_name, if already exist in array, then skip it
        3. check the second token to conc
        */

        token = strtok(buffer_ic50, ",");
        printf("%s\n", token); // testingAuto
        strcpy(tmp_drug_name, token);
        token = strtok(NULL, ",");
        printf("%s\n", token); // testingAuto
        strcat(tmp_drug_name, "_");
        strcat(tmp_drug_name, token);

        printf("%s\n", tmp_drug_name); // testingAuto
        addDrugData(&drug_name, drugsize, tmp_drug_name);
        conc[idx_conc++] = strtod(token, NULL);
        token = strtok(NULL, ",");
        // Check if there is wrong in here
        while (token != NULL) { // begin data tokenizing
            ic50[idx_ic50++] = strtod(token, NULL);
            printf("%s\n", token); // testingAuto
            token = strtok(NULL, ",");
        } // end data tokenizing
        sample_size++;
    } // end line reading

    fclose(fp_drugs);
    return sample_size;
}

int get_cvar_data_from_file(const char *file_name, unsigned int limit, double *cvar) {
    // buffer for writing in snprintf() function
    char buffer_cvar[255];
    FILE *fp_cvar;
    // cvar_t cvar;
    char *token;
    // std::array<double,18> temp_array;
    unsigned int idx;

    if ((fp_cvar = fopen(file_name, "r")) == NULL) {
        printf("Cannot open file %s\n",
               file_name);
    }
    idx = 0;
    int sample_size = 0;
    fgets(buffer_cvar, sizeof(buffer_cvar), fp_cvar);                                             // skip header
    while ((fgets(buffer_cvar, sizeof(buffer_cvar), fp_cvar) != NULL) && (sample_size < limit)) { // begin line reading
        token = strtok(buffer_cvar, ",");
        while (token != NULL) { // begin data tokenizing
            cvar[idx++] = strtod(token, NULL);
            token = strtok(NULL, ",");
        } // end data tokenizing
        // printf("\n");
        sample_size++;
        // cvar.push_back(temp_array);
    } // end line reading

    fclose(fp_cvar);
    return sample_size;
}

int get_init_data_from_file(const char *file_name, double *init_states) {
    // buffer for writing in snprintf() function
    char buffer_cache[1023];
    FILE *fp_cache;
    // cvar_t cvar;
    char *token;
    // std::array<double,18> temp_array;
    unsigned long idx;

    if ((fp_cache = fopen(file_name, "r")) == NULL) {
        printf("Cannot open file %s\n",
               file_name);
    }
    idx = 0;
    unsigned int sample_size = 0;
    fgets(buffer_cache, sizeof(buffer_cache), fp_cache); // skip header
    while ((fgets(buffer_cache, sizeof(buffer_cache), fp_cache) != NULL)) { // begin line reading
        token = strtok(buffer_cache, ",");
        while (token != NULL) { // begin data tokenizing
            init_states[idx++] = strtod(token, NULL);
            // if(idx < 82){
            //     printf("%d: %lf\n",idx-1,init_states[idx-1]);
            // }
            token = strtok(NULL, ",");
        } // end data tokenizing
        // printf("\n");
        sample_size++;
        // cvar.push_back(temp_array);
    } // end line reading

    fclose(fp_cache);
    return sample_size;
}

int exists(const char *fname) {
    FILE *file;
    if ((file = fopen(fname, "r"))) {
        fclose(file);
        return 1;
    }
    // fclose(file);
    return 0;
}

int check_IC50_content(const drug_t *ic50, const param_t *p_param) {
    if (ic50->size() == 0) {
        printf("Something problem with the IC50 file!\n");
        return 1;
    } else if (ic50->size() > 2000) {
        printf("Too much input! Maximum sample data is 2000!\n");
        return 2;
    } else if (p_param->pace_max < 750 && p_param->pace_max > 1000) {
        printf("Make sure the maximum pace is around 750 to 1000!\n");
        return 3;
    }
    // else if(mympi::size > ic50->size()){
    // 	printf("%s\n%s\n",
    //               "Overflow of MPI Process!",
    //               "Make sure MPI Size is less than or equal the number of sample");
    // 	return 4;
    // }
    else {
        return 0;
    }
}

int main(int argc, char **argv) {
    // enable real-time output in stdout
    setvbuf(stdout, NULL, _IONBF, 0);

    // NEW CODE STARTS HERE //
    // mycuda *thread_id;
    // hipMalloc(&thread_id, sizeof(mycuda));

    // input variables for cell simulation
    param_t *p_param, *d_p_param;
    p_param = new param_t();
    p_param->init();
    edison_assign_params(argc, argv, p_param);
    p_param->show_val();

    std::regex pattern("/([a-zA-Z0-9_\.]+)\.csv");
    std::smatch match;
    std::string fname = p_param->hill_file;
    std::regex_search(fname, match, pattern);
    
    printf("%s\n", match[1].str().c_str());

    double *ic50; // temporary
    double *cvar;
    double *conc;
    char **drug_name = nullptr;

    ic50 = (double *)malloc(14 * sample_limit * sizeof(double));
    // if (p_param->is_cvar == true) cvar = (double *)malloc(18 * sample_limit * sizeof(double));
    cvar = (double *)malloc(18 * sample_limit * sizeof(double));
    conc = (double *)malloc(sample_limit * sizeof(double));

    int Tomek_num_of_constants = 146;
    int Tomek_num_of_states = 42;
    int Tomek_num_of_algebraic = 199;
    int Tomek_num_of_rates = 42;

    //const double CONC = p_param->conc;

    //////// if we are in write time series mode (post processing) //////////
    if (p_param->is_time_series == 1 /*&& exists(p_param->cache_file) == 1 <- still unstable*/) {

        printf("Using cached initial state from previous result!!!! \n\n");

        const unsigned int datapoint_size = p_param->sampling_limit;
        double *cache;
        cache = (double *)malloc((Tomek_num_of_states + 2) * sample_limit * sizeof(double));

        double *d_ic50;
        double *d_conc;
        double *d_cvar;
        double *d_ALGEBRAIC;
        double *d_CONSTANTS;
        double *d_RATES;
        double *d_STATES;
        double *d_STATES_cache;
        double *d_mec_CONSTANTS, *d_mec_STATES, *d_mec_RATES, *d_mec_ALGEBRAIC;
        // actually not used but for now, this is only for satisfiying the GPU regulator parameters
        double *d_STATES_RESULT;
        double *d_all_states;

        double *time;
        double *dt;
        double *states;
        double *ical;
        double *inal;
        double *cai_result;
        double *ina;
        double *ito;
        double *ikr;
        double *iks;
        double *ik1;
        double *tension;
        cipa_t *temp_result, *cipa_result;

        int sample_size = get_IC50_data_from_file(p_param->hill_file, ic50, conc, drug_name);
        if (sample_size == 0)
            printf("Something problem with the IC50 file!\n");
        // else if(sample_size > 2000)
        //     printf("Too much input! Maximum sample data is 2000!\n");
        printf("Sample size: %d\n", sample_size);
        printf("Set GPU Number: %d\n", p_param->gpu_index);

        hipSetDevice(p_param->gpu_index);

        if (p_param->is_cvar == true) {
            int cvar_sample = get_cvar_data_from_file(p_param->cvar_file, sample_size, cvar);
            printf("Reading: %d Conductance Variability samples\n", cvar_sample);
        }

        printf("preparing GPU memory space \n");

        // char buffer_cvar[255];
        // snprintf(buffer_cvar, sizeof(buffer_cvar),
        // "./result/66_00.csv"
        // // "./drugs/optimized_pop_10k.csv"
        // );
        int cache_num = get_init_data_from_file(p_param->cache_file, cache);
        printf("Found cache for %d samples\n", cache_num);
        // note to self:
        // num of states+2 gave you at the very end of the file (pace number)
        // the very beginning -> the core number
        //   for (int z = 0; z <  Tomek_num_of_states; z++) {printf("%lf\n", cache[z+1]);}
        //   printf("\n");
        //   for (int z = 0; z <  Tomek_num_of_states; z++) {printf("%lf\n", cache[ 1*(Tomek_num_of_states+2) + (z+2)]);}
        //   printf("\n");
        //   for (int z = 0; z <  Tomek_num_of_states; z++) {printf("%lf\n", cache[ 2*(Tomek_num_of_states+2) + (z+3)]);}
        // return 0 ;

        hipMalloc(&d_ALGEBRAIC, Tomek_num_of_algebraic * sample_size * sizeof(double));
        hipMalloc(&d_CONSTANTS, Tomek_num_of_constants * sample_size * sizeof(double));
        hipMalloc(&d_RATES, Tomek_num_of_rates * sample_size * sizeof(double));
        hipMalloc(&d_STATES, Tomek_num_of_states * sample_size * sizeof(double));
        hipMalloc(&d_STATES_cache, (Tomek_num_of_states + 2) * sample_size * sizeof(double));
        hipMalloc(&d_mec_ALGEBRAIC, 24 * sample_size * sizeof(double));
        hipMalloc(&d_mec_CONSTANTS, 29 * sample_size * sizeof(double));
        hipMalloc(&d_mec_RATES, 7 * sample_size * sizeof(double));
        hipMalloc(&d_mec_STATES, 7 * sample_size * sizeof(double));

        hipMalloc(&d_p_param, sizeof(param_t));

        // prep for 1 cycle plus a bit (7000 * sample_size)
        hipMalloc(&temp_result, sample_size * sizeof(cipa_t));
        hipMalloc(&cipa_result, sample_size * sizeof(cipa_t));

        hipMalloc(&time, sample_size * datapoint_size * sizeof(double));
        hipMalloc(&dt, sample_size * datapoint_size * sizeof(double));
        hipMalloc(&states, sample_size * datapoint_size * sizeof(double));
        hipMalloc(&ical, sample_size * datapoint_size * sizeof(double));
        hipMalloc(&inal, sample_size * datapoint_size * sizeof(double));
        hipMalloc(&cai_result, sample_size * datapoint_size * sizeof(double));
        hipMalloc(&ina, sample_size * datapoint_size * sizeof(double));
        hipMalloc(&ito, sample_size * datapoint_size * sizeof(double));
        hipMalloc(&ikr, sample_size * datapoint_size * sizeof(double));
        hipMalloc(&iks, sample_size * datapoint_size * sizeof(double));
        hipMalloc(&ik1, sample_size * datapoint_size * sizeof(double));
        hipMalloc(&tension, sample_size * datapoint_size * sizeof(double));
        // hipMalloc(&d_STATES_RESULT, (Tomek_num_of_states+1) * sample_size * sizeof(double));
        // hipMalloc(&d_all_states, Tomek_num_of_states * sample_size * p_param->find_steepest_start * sizeof(double));

        printf("Copying sample files to GPU memory space \n");
        hipMalloc(&d_ic50, sample_size * 14 * sizeof(double));
        hipMalloc(&d_cvar, sample_size * 18 * sizeof(double));
        hipMalloc(&d_conc, sample_size * sizeof(double));
        hipMemcpy(d_STATES_cache, cache, (Tomek_num_of_states + 2) * sample_size * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_ic50, ic50, sample_size * 14 * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_cvar, cvar, sample_size * 18 * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_conc, conc, sample_size * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_p_param, p_param, sizeof(param_t), hipMemcpyHostToDevice);

        // // Get the maximum number of active blocks per multiprocessor
        // hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocks, do_drug_sim_analytical, threadsPerBlock);

        // // Calculate the total number of blocks
        // int numTotalBlocks = numBlocks * cudaDeviceGetMultiprocessorCount();

        tic();
        printf("Timer started, doing simulation.... \n\n\nGPU Usage at this moment: \n");
        int thread = 32;
        int block = (sample_size + thread - 1) / thread;
        // int block = (sample_size + thread - 1) / thread;
        if (gpu_check(15 * sample_size * sizeof(double) + sizeof(param_t)) == 1) {
            printf("GPU memory insufficient!\n");
            return 0;
        }
        printf("Sample size: %d\n", sample_size);
        hipSetDevice(p_param->gpu_index);
        printf("\n   Configuration: \n\n\tblock\t||\tthread\n---------------------------------------\n  \t%d\t||\t%d\n\n\n", block, thread);
        // initscr();
        // printf("[____________________________________________________________________________________________________]  0.00 %% \n");

        kernel_DrugSimulation<<<block, thread>>>(d_ic50, d_cvar, d_conc, d_CONSTANTS, d_STATES, d_STATES_cache, d_RATES, d_ALGEBRAIC,
                                                 d_mec_CONSTANTS, d_mec_STATES, d_mec_RATES, d_mec_ALGEBRAIC,
                                                 d_STATES_RESULT, d_all_states,
                                                 time, states, dt, cai_result,
                                                 ina, inal,
                                                 ical, ito,
                                                 ikr, iks,
                                                 ik1, tension,
                                                 sample_size,
                                                 temp_result, cipa_result,
                                                 d_p_param);
        // block per grid, threads per block
        // endwin();

        hipDeviceSynchronize();

        printf("allocating memory for computation result in the CPU, malloc style \n");
        double *h_states, *h_time, *h_dt, *h_ical, *h_inal, *h_cai_result, *h_ina, *h_ito, *h_ikr, *h_iks, *h_ik1, *h_tension;
        cipa_t *h_cipa_result;

        h_states = (double *)malloc(datapoint_size * sample_size * sizeof(double));
        printf("...allocated for STATES, \n");
        h_time = (double *)malloc(datapoint_size * sample_size * sizeof(double));
        printf("...allocated for time, \n");
        h_dt = (double *)malloc(datapoint_size * sample_size * sizeof(double));
        printf("...allocated for dt, \n");
        h_cai_result = (double *)malloc(datapoint_size * sample_size * sizeof(double));
        printf("...allocated for Cai, \n");
        h_ina = (double *)malloc(datapoint_size * sample_size * sizeof(double));
        printf("...allocated for iNa, \n");
        h_ito = (double *)malloc(datapoint_size * sample_size * sizeof(double));
        printf("...allocated for ito, \n");
        h_ikr = (double *)malloc(datapoint_size * sample_size * sizeof(double));
        printf("...allocated for ikr, \n");
        h_iks = (double *)malloc(datapoint_size * sample_size * sizeof(double));
        printf("...allocated for iks, \n");
        h_ik1 = (double *)malloc(datapoint_size * sample_size * sizeof(double));
        printf("...allocated for ik1, \n");
        h_ical = (double *)malloc(datapoint_size * sample_size * sizeof(double));
        printf("...allocated for ICaL, \n");
        h_inal = (double *)malloc(datapoint_size * sample_size * sizeof(double));
        h_tension = (double *)malloc(datapoint_size * sample_size * sizeof(double));
        h_cipa_result = (cipa_t *)malloc(sample_size * sizeof(cipa_t));
        printf("...allocating for INaL and postprocessing, all set!\n");

        ////// copy the data back to CPU, and write them into file ////////
        printf("copying the data back to the CPU \n");

        hipMemcpy(h_states, states, sample_size * datapoint_size * sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(h_time, time, sample_size * datapoint_size * sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(h_dt, dt, sample_size * datapoint_size * sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(h_ical, ical, sample_size * datapoint_size * sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(h_inal, inal, sample_size * datapoint_size * sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(h_cai_result, cai_result, sample_size * datapoint_size * sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(h_ina, ina, sample_size * datapoint_size * sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(h_ito, ito, sample_size * datapoint_size * sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(h_ikr, ikr, sample_size * datapoint_size * sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(h_iks, iks, sample_size * datapoint_size * sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(h_ik1, ik1, sample_size * datapoint_size * sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(h_tension, tension, sample_size * datapoint_size * sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(h_cipa_result, cipa_result, sample_size * sizeof(cipa_t), hipMemcpyDeviceToHost);

        hipFree(d_ALGEBRAIC);
        hipFree(d_CONSTANTS);
        hipFree(d_RATES);
        hipFree(d_STATES);
        hipFree(d_mec_ALGEBRAIC);
        hipFree(d_mec_CONSTANTS);
        hipFree(d_mec_RATES);
        hipFree(d_mec_STATES);
        hipFree(d_p_param);
        hipFree(temp_result);
        hipFree(cipa_result);
        hipFree(d_STATES_RESULT);
        hipFree(d_ic50);
        hipFree(d_cvar);
        hipFree(d_conc);
        hipFree(time);
        hipFree(dt);
        hipFree(states);
        hipFree(ical);
        hipFree(inal);
        hipFree(cai_result);
        hipFree(ina);
        hipFree(ito);
        hipFree(ikr);
        hipFree(iks);
        hipFree(ik1);
        hipFree(tension);
    
        FILE *writer;
        int check;
        bool folder_created = false;

        printf("writing to file... \n");
        // sample loop
        for (int sample_id = 0; sample_id < sample_size; sample_id++) {
            // printf("writing sample %d... \n",sample_id);
            char sample_str[ENOUGH];
            char conc_str[ENOUGH];
            char filename[500] = "./result/post_";
            sprintf(sample_str, "%d", sample_id);
            //sprintf(conc_str, "%.2f", conc[sample_id]);
            strcat(filename, match[1].str().c_str());
            strcat(filename, "/");
            if (folder_created == false) {
                check = mkdir(filename, 0777);
                // check if directory is created or not
                if (!check) {
                    printf("Directory created\n");
                } else {
                    printf("Unable to create directory, or the folder is already created, relax mate...\n");
                }
                folder_created = true;
            }

            strcat(filename, sample_str);
            strcat(filename, "_pace.csv");

            writer = fopen(filename, "w");
            fprintf(writer, "Time,Vm,dVm/dt,Cai,INa,INaL,ICaL,IKs,IKr,IK1,Ito,Tension\n");
            for (int datapoint = 1; datapoint < datapoint_size; datapoint++) {
                if (h_time[ sample_id + (datapoint * sample_size)] == 0.0) {break;}
                fprintf(writer, "%lf,%lf,%lf,%lf,%lf,%lf,%lf,%lf,%lf,%lf,%lf,%lf\n", // change this into string, or limit the decimal accuracy, so we can decrease filesize
                        h_time[sample_id + (datapoint * sample_size)],
                        h_states[sample_id + (datapoint * sample_size)],
                        h_dt[sample_id + (datapoint * sample_size)],
                        h_cai_result[sample_id + (datapoint * sample_size)],

                        h_ina[sample_id + (datapoint * sample_size)],
                        h_inal[sample_id + (datapoint * sample_size)],

                        h_ical[sample_id + (datapoint * sample_size)],
                        h_iks[sample_id + (datapoint * sample_size)],

                        h_ikr[sample_id + (datapoint * sample_size)],
                        h_ik1[sample_id + (datapoint * sample_size)],

                        h_ito[sample_id + (datapoint * sample_size)],
                        h_tension[sample_id + (datapoint * sample_size)]);
            }
            fclose(writer);
        }

        printf("writing each biomarkers value... \n");
        // sample loop
        // char conc_str[ENOUGH];
        char filename[500] = "./result/post_";
        // sprintf(sample_str, "%d", sample_id);
        // sprintf(conc_str, "%.2f", conc[sample_id]);
        strcat(filename, match[1].str().c_str());
        strcat(filename, "/");
        // printf("creating %s... \n", filename);
        if (folder_created == false) {
            check = mkdir(filename, 0777);
            // check if directory is created or not
            if (!check) {
                printf("Directory created\n");
            } else {
                printf("Unable to create directory, or the folder is already created, relax mate...\n");
            }
            folder_created = true;
        }

        // strcat(filename,sample_str);
        strcat(filename, "_biomarkers.csv");

        writer = fopen(filename, "a");

        fprintf(writer, "sample,qnet,inal_auc,ical_auc,apd90,apd50,apd_tri,cad90,cad50,cad_tri,dvmdt_repol,vm_peak,vm_valley,vm_dia,ca_peak,ca_valley,ca_dia\n");
        for (int sample_id = 0; sample_id < sample_size; sample_id++) {
            // printf("writing sample %d... \n",sample_id);

            fprintf(writer, "%d,%lf,%lf,%lf,%lf,%lf,%lf,%lf,%lf,%lf,%lf,%lf,%lf,%lf,%lf,%lf,%lf\n", // change this into string, or limit the decimal accuracy, so we can decrease filesize
                    sample_id,
                    h_cipa_result[sample_id].qnet,
                    h_cipa_result[sample_id].inal_auc,
                    h_cipa_result[sample_id].ical_auc,

                    h_cipa_result[sample_id].apd90,
                    h_cipa_result[sample_id].apd50,
                    h_cipa_result[sample_id].apd90 - h_cipa_result[sample_id].apd50,

                    h_cipa_result[sample_id].cad90,
                    h_cipa_result[sample_id].cad50,
                    h_cipa_result[sample_id].cad90 - h_cipa_result[sample_id].cad50,

                    h_cipa_result[sample_id].dvmdt_repol,
                    h_cipa_result[sample_id].vm_peak,
                    h_cipa_result[sample_id].vm_valley,
                    h_cipa_result[sample_id].vm_dia,

                    h_cipa_result[sample_id].ca_peak,
                    h_cipa_result[sample_id].ca_valley,
                    h_cipa_result[sample_id].ca_dia

                    //      temp_result[sample_id].qnet = 0.;
                    // temp_result[sample_id].inal_auc = 0.;
                    // temp_result[sample_id].ical_auc = 0.;

                    // temp_result[sample_id].dvmdt_repol = -999;
                    // temp_result[sample_id].dvmdt_max = -999;
                    // temp_result[sample_id].vm_peak = -999;
                    // temp_result[sample_id].vm_valley = d_STATES[(sample_id * Tomek_num_of_states) +V];
                    // temp_result[sample_id].vm_dia = -999;

                    // temp_result[sample_id].apd90 = 0.;
                    // temp_result[sample_id].apd50 = 0.;
                    // temp_result[sample_id].ca_peak = -999;
                    // temp_result[sample_id].ca_valley = d_STATES[(sample_id * Tomek_num_of_states) +cai];
                    // temp_result[sample_id].ca_dia = -999;
                    // temp_result[sample_id].cad90 = 0.;
                    // temp_result[sample_id].cad50 = 0.;
            );
        }
        fclose(writer);

        toc();

        return 0;
    }
    return 0;
    ////////// find cache mode (in silico code) //////////
    
}
